
#include <hip/hip_runtime.h>
/******************************************************************************
 *cr
 *cr            (C) Copyright 2010 The Board of Trustees of the
 *cr                        University of Illinois
 *cr                         All Rights Reserved
 *cr
 ******************************************************************************/

// Define your kernels in this file you may use more than one kernel if you
// need to

// INSERT KERNEL(S) HERE

__global__ void hist_kernel(unsigned int* input, unsigned int* bins, unsigned int num_elements, unsigned int num_bins) {

    extern __shared__ unsigned int hist[];

    int i = 0;
    while(i * blockDim.x + threadIdx.x < num_bins){
	hist[i * blockDim.x + threadIdx.x] = 0;
	i++;
    }
    __syncthreads();

    i = threadIdx.x + blockDim.x * blockIdx.x;

    int stride = blockDim.x * gridDim.x;
    int val = 0;
    while(val * stride + i < num_elements) {
	atomicAdd(&(hist[input[val * stride + i]]), 1);
	val++;
    }
    __syncthreads();
    
    val = 0;
    while(val * blockDim.x + threadIdx.x < num_bins) {
	atomicAdd(&bins[val * blockDim.x + threadIdx.x], hist[val * blockDim.x + threadIdx.x]);
  	val++;
    }
}

/******************************************************************************
Setup and invoke your kernel(s) in this function. You may also allocate more
GPU memory if you need to
*******************************************************************************/
void histogram(unsigned int* input, unsigned int* bins, unsigned int num_elements, unsigned int num_bins) {

    // INSERT CODE HERE
    const unsigned int BLOCK_SIZE = 256;

    dim3 DimGrid( (num_elements-1)/BLOCK_SIZE + 1, 1, 1);
    dim3 DimBlock(BLOCK_SIZE, 1, 1);
//    dim3 DimGrid(30,1,1);
//    dim3 DimBlock(32,1,1);
    printf("Number of blocks: %d\n", (num_elements-1)/BLOCK_SIZE + 1);

    hist_kernel<<<DimGrid, DimBlock, num_bins*sizeof(unsigned int)>>>(input, bins, num_elements, num_bins);

}
